#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

__global__ void vectorAddition(int rows, int cols, float* a, float* b, float* c) {
    // get global thread index
    int i = threadIdx.x + threadIdx.y * blockDim.x;

    // vector addition per thread
    if (idx < rows * cols) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int rows = 1024;
    int cols = 1024;

    // init input/output matrices in host
    float* host_a = (float*)malloc(rows * cols * sizeof(float));
    float* host_b = (float*)malloc(rows * cols * sizeof(float));
    float* host_c = (float*)malloc(rows * cols * sizeof(float));

    srand(5566);
    for (int i = 0; i < rows * cols; i++) {
        host_a[i] = (float)rand() / RAND_MAX;
        host_b[i] = (float)rand() / RAND_MAX;
    }
    
    // copy input/output matrices to device
    float* device_a;
    float* device_b;
    float* device_c;

    hipMalloc(&device_a, rows * cols * sizeof(float));
    hipMalloc(&device_b, rows * cols * sizeof(float));

    hipMemcpy(device_a, host_a, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // vector addition
    int threadsPerBlock = 16;
    int blocksPerGrid = (rows + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(rows, cols, device_a, device_b, device_c);
    hipDeviceSynchronize(); // wait for the kernel to finish

    // copy output matrices to host
    hipMemcpy(host_c, device_c, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // show result
    for (int i = 0; i < rows * cols; i++) {
        printf("%f + %f = %f\n", host_a[i], host_b[i], host_c[i]);
    }

    // clean up
    free(host_a);
    free(host_b);
    free(host_c);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;
}